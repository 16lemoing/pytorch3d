#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <float.h>
#include <math.h>
#include <tuple>
#include "rasterize_coarse/bitmask.cuh"
#include "rasterize_points/rasterization_utils.cuh"
#include "utils/float_math.cuh"
#include "utils/geometry_utils.cuh" // For kEpsilon -- gross

// Get the xyz coordinates of the three vertices for the face given by the
// index face_idx into face_verts.
__device__ thrust::tuple<float3, float3, float3> GetSingleFaceVerts(
    const float* face_verts,
    int face_idx) {
  const float x0 = face_verts[face_idx * 9 + 0];
  const float y0 = face_verts[face_idx * 9 + 1];
  const float z0 = face_verts[face_idx * 9 + 2];
  const float x1 = face_verts[face_idx * 9 + 3];
  const float y1 = face_verts[face_idx * 9 + 4];
  const float z1 = face_verts[face_idx * 9 + 5];
  const float x2 = face_verts[face_idx * 9 + 6];
  const float y2 = face_verts[face_idx * 9 + 7];
  const float z2 = face_verts[face_idx * 9 + 8];

  const float3 v0xyz = make_float3(x0, y0, z0);
  const float3 v1xyz = make_float3(x1, y1, z1);
  const float3 v2xyz = make_float3(x2, y2, z2);

  return thrust::make_tuple(v0xyz, v1xyz, v2xyz);
}

__global__ void RasterizeMeshesCoarseCudaKernel(
    const float* face_verts,
    const int64_t* mesh_to_face_first_idx,
    const int64_t* num_faces_per_mesh,
    const float blur_radius,
    const int N,
    const int F,
    const int H,
    const int W,
    const int bin_size,
    const int chunk_size,
    const int max_faces_per_bin,
    int* faces_per_bin,
    int* bin_faces) {
  extern __shared__ char sbuf[];
  const int M = max_faces_per_bin;
  // Integer divide round up
  const int num_bins_x = 1 + (W - 1) / bin_size;
  const int num_bins_y = 1 + (H - 1) / bin_size;

  // NDC range depends on the ratio of W/H
  // The shorter side from (H, W) is given an NDC range of 2.0 and
  // the other side is scaled by the ratio of H:W.
  const float NDC_x_half_range = NonSquareNdcRange(W, H) / 2.0f;
  const float NDC_y_half_range = NonSquareNdcRange(H, W) / 2.0f;

  // Size of half a pixel in NDC units is the NDC half range
  // divided by the corresponding image dimension
  const float half_pix_x = NDC_x_half_range / W;
  const float half_pix_y = NDC_y_half_range / H;

  // This is a boolean array of shape (num_bins_y, num_bins_x, chunk_size)
  // stored in shared memory that will track whether each point in the chunk
  // falls into each bin of the image.
  BitMask binmask((unsigned int*)sbuf, num_bins_y, num_bins_x, chunk_size);

  // Have each block handle a chunk of faces
  const int chunks_per_batch = 1 + (F - 1) / chunk_size;
  const int num_chunks = N * chunks_per_batch;

  for (int chunk = blockIdx.x; chunk < num_chunks; chunk += gridDim.x) {
    const int batch_idx = chunk / chunks_per_batch; // batch index
    const int chunk_idx = chunk % chunks_per_batch;
    const int face_start_idx = chunk_idx * chunk_size;

    binmask.block_clear();
    const int64_t mesh_face_start_idx = mesh_to_face_first_idx[batch_idx];
    const int64_t mesh_face_stop_idx =
        mesh_face_start_idx + num_faces_per_mesh[batch_idx];

    // Have each thread handle a different face within the chunk
    for (int f = threadIdx.x; f < chunk_size; f += blockDim.x) {
      const int f_idx = face_start_idx + f;

      // Check if face index corresponds to the mesh in the batch given by
      // batch_idx
      if (f_idx >= mesh_face_stop_idx || f_idx < mesh_face_start_idx) {
        continue;
      }

      // Get xyz coordinates of the three face vertices.
      const auto v012 = GetSingleFaceVerts(face_verts, f_idx);
      const float3 v0 = thrust::get<0>(v012);
      const float3 v1 = thrust::get<1>(v012);
      const float3 v2 = thrust::get<2>(v012);

      // Compute screen-space bbox for the triangle expanded by blur.
      float xmin = FloatMin3(v0.x, v1.x, v2.x) - sqrt(blur_radius);
      float ymin = FloatMin3(v0.y, v1.y, v2.y) - sqrt(blur_radius);
      float xmax = FloatMax3(v0.x, v1.x, v2.x) + sqrt(blur_radius);
      float ymax = FloatMax3(v0.y, v1.y, v2.y) + sqrt(blur_radius);
      float zmin = FloatMin3(v0.z, v1.z, v2.z);

      // Faces with at least one vertex behind the camera won't render
      // correctly and should be removed or clipped before calling the
      // rasterizer
      if (zmin < kEpsilon) {
        continue;
      }

      // Brute-force search over all bins; TODO(T54294966) something smarter.
      for (int by = 0; by < num_bins_y; ++by) {
        // Y coordinate of the top and bottom of the bin.
        // PixToNdc gives the location of the center of each pixel, so we
        // need to add/subtract a half pixel to get the true extent of the bin.
        // Reverse ordering of Y axis so that +Y is upwards in the image.
        const float bin_y_min =
            PixToNonSquareNdc(by * bin_size, H, W) - half_pix_y;
        const float bin_y_max =
            PixToNonSquareNdc((by + 1) * bin_size - 1, H, W) + half_pix_y;
        const bool y_overlap = (ymin <= bin_y_max) && (bin_y_min < ymax);

        for (int bx = 0; bx < num_bins_x; ++bx) {
          // X coordinate of the left and right of the bin.
          // Reverse ordering of x axis so that +X is left.
          const float bin_x_max =
              PixToNonSquareNdc((bx + 1) * bin_size - 1, W, H) + half_pix_x;
          const float bin_x_min =
              PixToNonSquareNdc(bx * bin_size, W, H) - half_pix_x;

          const bool x_overlap = (xmin <= bin_x_max) && (bin_x_min < xmax);
          if (y_overlap && x_overlap) {
            binmask.set(by, bx, f);
          }
        }
      }
    }
    __syncthreads();
    // Now we have processed every face in the current chunk. We need to
    // count the number of faces in each bin so we can write the indices
    // out to global memory. We have each thread handle a different bin.
    for (int byx = threadIdx.x; byx < num_bins_y * num_bins_x;
         byx += blockDim.x) {
      const int by = byx / num_bins_x;
      const int bx = byx % num_bins_x;
      const int count = binmask.count(by, bx);
      const int faces_per_bin_idx =
          batch_idx * num_bins_y * num_bins_x + by * num_bins_x + bx;

      // This atomically increments the (global) number of faces found
      // in the current bin, and gets the previous value of the counter;
      // this effectively allocates space in the bin_faces array for the
      // faces in the current chunk that fall into this bin.
      const int start = atomicAdd(faces_per_bin + faces_per_bin_idx, count);

      // Now loop over the binmask and write the active bits for this bin
      // out to bin_faces.
      int next_idx = batch_idx * num_bins_y * num_bins_x * M +
          by * num_bins_x * M + bx * M + start;
      for (int f = 0; f < chunk_size; ++f) {
        if (binmask.get(by, bx, f)) {
          // TODO(T54296346) find the correct method for handling errors in
          // CUDA. Throw an error if num_faces_per_bin > max_faces_per_bin.
          // Either decrease bin size or increase max_faces_per_bin
          bin_faces[next_idx] = face_start_idx + f;
          next_idx++;
        }
      }
    }
    __syncthreads();
  }
}

__global__ void RasterizePointsCoarseCudaKernel(
    const float* points, // (P, 3)
    const int64_t* cloud_to_packed_first_idx, // (N)
    const int64_t* num_points_per_cloud, // (N)
    const float* radius,
    const int N,
    const int P,
    const int H,
    const int W,
    const int bin_size,
    const int chunk_size,
    const int max_points_per_bin,
    int* points_per_bin,
    int* bin_points) {
  extern __shared__ char sbuf[];
  const int M = max_points_per_bin;

  // Integer divide round up
  const int num_bins_x = 1 + (W - 1) / bin_size;
  const int num_bins_y = 1 + (H - 1) / bin_size;

  // NDC range depends on the ratio of W/H
  // The shorter side from (H, W) is given an NDC range of 2.0 and
  // the other side is scaled by the ratio of H:W.
  const float NDC_x_half_range = NonSquareNdcRange(W, H) / 2.0f;
  const float NDC_y_half_range = NonSquareNdcRange(H, W) / 2.0f;

  // Size of half a pixel in NDC units is the NDC half range
  // divided by the corresponding image dimension
  const float half_pix_x = NDC_x_half_range / W;
  const float half_pix_y = NDC_y_half_range / H;

  // This is a boolean array of shape (num_bins_y, num_bins_x, chunk_size)
  // stored in shared memory that will track whether each point in the chunk
  // falls into each bin of the image.
  BitMask binmask((unsigned int*)sbuf, num_bins_y, num_bins_x, chunk_size);

  // Have each block handle a chunk of points and build a 3D bitmask in
  // shared memory to mark which points hit which bins.  In this first phase,
  // each thread processes one point at a time. After processing the chunk,
  // one thread is assigned per bin, and the thread counts and writes the
  // points for the bin out to global memory.
  const int chunks_per_batch = 1 + (P - 1) / chunk_size;
  const int num_chunks = N * chunks_per_batch;
  for (int chunk = blockIdx.x; chunk < num_chunks; chunk += gridDim.x) {
    const int batch_idx = chunk / chunks_per_batch;
    const int chunk_idx = chunk % chunks_per_batch;
    const int point_start_idx = chunk_idx * chunk_size;

    binmask.block_clear();

    // Using the batch index of the thread get the start and stop
    // indices for the points.
    const int64_t cloud_point_start_idx = cloud_to_packed_first_idx[batch_idx];
    const int64_t cloud_point_stop_idx =
        cloud_point_start_idx + num_points_per_cloud[batch_idx];

    // Have each thread handle a different point within the chunk
    for (int p = threadIdx.x; p < chunk_size; p += blockDim.x) {
      const int p_idx = point_start_idx + p;

      // Check if point index corresponds to the cloud in the batch given by
      // batch_idx.
      if (p_idx >= cloud_point_stop_idx || p_idx < cloud_point_start_idx) {
        continue;
      }

      const float px = points[p_idx * 3 + 0];
      const float py = points[p_idx * 3 + 1];
      const float pz = points[p_idx * 3 + 2];
      const float p_radius = radius[p_idx];
      if (pz < 0)
        continue; // Don't render points behind the camera.
      const float px0 = px - p_radius;
      const float px1 = px + p_radius;
      const float py0 = py - p_radius;
      const float py1 = py + p_radius;

      // Brute-force search over all bins; TODO something smarter?
      // For example we could compute the exact bin where the point falls,
      // then check neighboring bins. This way we wouldn't have to check
      // all bins (however then we might have more warp divergence?)
      for (int by = 0; by < num_bins_y; ++by) {
        // Get y extent for the bin. PixToNonSquareNdc gives us the location of
        // the center of each pixel, so we need to add/subtract a half
        // pixel to get the true extent of the bin.
        const float by0 = PixToNonSquareNdc(by * bin_size, H, W) - half_pix_y;
        const float by1 =
            PixToNonSquareNdc((by + 1) * bin_size - 1, H, W) + half_pix_y;
        const bool y_overlap = (py0 <= by1) && (by0 <= py1);

        if (!y_overlap) {
          continue;
        }
        for (int bx = 0; bx < num_bins_x; ++bx) {
          // Get x extent for the bin; again we need to adjust the
          // output of PixToNonSquareNdc by half a pixel.
          const float bx0 = PixToNonSquareNdc(bx * bin_size, W, H) - half_pix_x;
          const float bx1 =
              PixToNonSquareNdc((bx + 1) * bin_size - 1, W, H) + half_pix_x;
          const bool x_overlap = (px0 <= bx1) && (bx0 <= px1);

          if (x_overlap) {
            binmask.set(by, bx, p);
          }
        }
      }
    }
    __syncthreads();
    // Now we have processed every point in the current chunk. We need to
    // count the number of points in each bin so we can write the indices
    // out to global memory. We have each thread handle a different bin.
    for (int byx = threadIdx.x; byx < num_bins_y * num_bins_x;
         byx += blockDim.x) {
      const int by = byx / num_bins_x;
      const int bx = byx % num_bins_x;
      const int count = binmask.count(by, bx);
      const int points_per_bin_idx =
          batch_idx * num_bins_y * num_bins_x + by * num_bins_x + bx;

      // This atomically increments the (global) number of points found
      // in the current bin, and gets the previous value of the counter;
      // this effectively allocates space in the bin_points array for the
      // points in the current chunk that fall into this bin.
      const int start = atomicAdd(points_per_bin + points_per_bin_idx, count);

      // Now loop over the binmask and write the active bits for this bin
      // out to bin_points.
      int next_idx = batch_idx * num_bins_y * num_bins_x * M +
          by * num_bins_x * M + bx * M + start;
      for (int p = 0; p < chunk_size; ++p) {
        if (binmask.get(by, bx, p)) {
          // TODO: Throw an error if next_idx >= M -- this means that
          // we got more than max_points_per_bin in this bin
          // TODO: check if atomicAdd is needed in line 265.
          bin_points[next_idx] = point_start_idx + p;
          next_idx++;
        }
      }
    }
    __syncthreads();
  }
}

at::Tensor RasterizeMeshesCoarseCuda(
    const at::Tensor& face_verts,
    const at::Tensor& mesh_to_face_first_idx,
    const at::Tensor& num_faces_per_mesh,
    const std::tuple<int, int> image_size,
    const float blur_radius,
    const int bin_size,
    const int max_faces_per_bin) {
  TORCH_CHECK(
      face_verts.ndimension() == 3 && face_verts.size(1) == 3 &&
          face_verts.size(2) == 3,
      "face_verts must have dimensions (num_faces, 3, 3)");

  // Check inputs are on the same device
  at::TensorArg face_verts_t{face_verts, "face_verts", 1},
      mesh_to_face_first_idx_t{
          mesh_to_face_first_idx, "mesh_to_face_first_idx", 2},
      num_faces_per_mesh_t{num_faces_per_mesh, "num_faces_per_mesh", 3};
  at::CheckedFrom c = "RasterizeMeshesCoarseCuda";
  at::checkAllSameGPU(
      c, {face_verts_t, mesh_to_face_first_idx_t, num_faces_per_mesh_t});

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(face_verts.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const int H = std::get<0>(image_size);
  const int W = std::get<1>(image_size);

  const int F = face_verts.size(0);
  const int N = num_faces_per_mesh.size(0);
  const int M = max_faces_per_bin;

  // Integer divide round up.
  const int num_bins_y = 1 + (H - 1) / bin_size;
  const int num_bins_x = 1 + (W - 1) / bin_size;

  if (num_bins_y >= kMaxItemsPerBin || num_bins_x >= kMaxItemsPerBin) {
    std::stringstream ss;
    ss << "In Coarse Rasterizer got num_bins_y: " << num_bins_y
       << ", num_bins_x: " << num_bins_x << ", "
       << "; that's too many!";
    AT_ERROR(ss.str());
  }
  auto opts = num_faces_per_mesh.options().dtype(at::kInt);
  at::Tensor faces_per_bin = at::zeros({N, num_bins_y, num_bins_x}, opts);
  at::Tensor bin_faces = at::full({N, num_bins_y, num_bins_x, M}, -1, opts);

  if (bin_faces.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return bin_faces;
  }

  const int chunk_size = 512;
  const size_t shared_size = num_bins_y * num_bins_x * chunk_size / 8;
  const size_t blocks = 64;
  const size_t threads = 512;

  RasterizeMeshesCoarseCudaKernel<<<blocks, threads, shared_size, stream>>>(
      face_verts.contiguous().data_ptr<float>(),
      mesh_to_face_first_idx.contiguous().data_ptr<int64_t>(),
      num_faces_per_mesh.contiguous().data_ptr<int64_t>(),
      blur_radius,
      N,
      F,
      H,
      W,
      bin_size,
      chunk_size,
      M,
      faces_per_bin.data_ptr<int32_t>(),
      bin_faces.data_ptr<int32_t>());

  AT_CUDA_CHECK(hipGetLastError());
  return bin_faces;
}

at::Tensor RasterizePointsCoarseCuda(
    const at::Tensor& points, // (P, 3)
    const at::Tensor& cloud_to_packed_first_idx, // (N)
    const at::Tensor& num_points_per_cloud, // (N)
    const std::tuple<int, int> image_size,
    const at::Tensor& radius,
    const int bin_size,
    const int max_points_per_bin) {
  TORCH_CHECK(
      points.ndimension() == 2 && points.size(1) == 3,
      "points must have dimensions (num_points, 3)");

  // Check inputs are on the same device
  at::TensorArg points_t{points, "points", 1},
      cloud_to_packed_first_idx_t{
          cloud_to_packed_first_idx, "cloud_to_packed_first_idx", 2},
      num_points_per_cloud_t{num_points_per_cloud, "num_points_per_cloud", 3};
  at::CheckedFrom c = "RasterizePointsCoarseCuda";
  at::checkAllSameGPU(
      c, {points_t, cloud_to_packed_first_idx_t, num_points_per_cloud_t});

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(points.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const int H = std::get<0>(image_size);
  const int W = std::get<1>(image_size);

  const int P = points.size(0);
  const int N = num_points_per_cloud.size(0);
  const int M = max_points_per_bin;

  // Integer divide round up.
  const int num_bins_y = 1 + (H - 1) / bin_size;
  const int num_bins_x = 1 + (W - 1) / bin_size;

  if (num_bins_y >= kMaxItemsPerBin || num_bins_x >= kMaxItemsPerBin) {
    // Make sure we do not use too much shared memory.
    std::stringstream ss;
    ss << "In Coarse Rasterizer got num_bins_y: " << num_bins_y
       << ", num_bins_x: " << num_bins_x << ", "
       << "; that's too many!";
    AT_ERROR(ss.str());
  }
  auto opts = num_points_per_cloud.options().dtype(at::kInt);
  at::Tensor points_per_bin = at::zeros({N, num_bins_y, num_bins_x}, opts);
  at::Tensor bin_points = at::full({N, num_bins_y, num_bins_x, M}, -1, opts);

  if (bin_points.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return bin_points;
  }

  const int chunk_size = 512;
  const size_t shared_size = num_bins_y * num_bins_x * chunk_size / 8;
  const size_t blocks = 64;
  const size_t threads = 512;

  RasterizePointsCoarseCudaKernel<<<blocks, threads, shared_size, stream>>>(
      points.contiguous().data_ptr<float>(),
      cloud_to_packed_first_idx.contiguous().data_ptr<int64_t>(),
      num_points_per_cloud.contiguous().data_ptr<int64_t>(),
      radius.contiguous().data_ptr<float>(),
      N,
      P,
      H,
      W,
      bin_size,
      chunk_size,
      M,
      points_per_bin.contiguous().data_ptr<int32_t>(),
      bin_points.contiguous().data_ptr<int32_t>());

  AT_CUDA_CHECK(hipGetLastError());
  return bin_points;
}
