#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 * All rights reserved.
 *
 * This source code is licensed under the BSD-style license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <float.h>
#include <math.h>
#include <tuple>
#include "rasterize_coarse/bitmask.cuh"
#include "rasterize_points/rasterization_utils.cuh"
#include "utils/float_math.cuh"
#include "utils/geometry_utils.cuh" // For kEpsilon -- gross

__global__ void TriangleBoundingBoxKernel(
    const float* face_verts, // (F, 3, 3)
    const int F,
    const float blur_radius,
    float* bboxes, // (4, F)
    bool* skip_face) { // (F,)
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  const int num_threads = blockDim.x * gridDim.x;
  const float sqrt_radius = sqrt(blur_radius);
  for (int f = tid; f < F; f += num_threads) {
    const float v0x = face_verts[f * 9 + 0 * 3 + 0];
    const float v0y = face_verts[f * 9 + 0 * 3 + 1];
    const float v0z = face_verts[f * 9 + 0 * 3 + 2];
    const float v1x = face_verts[f * 9 + 1 * 3 + 0];
    const float v1y = face_verts[f * 9 + 1 * 3 + 1];
    const float v1z = face_verts[f * 9 + 1 * 3 + 2];
    const float v2x = face_verts[f * 9 + 2 * 3 + 0];
    const float v2y = face_verts[f * 9 + 2 * 3 + 1];
    const float v2z = face_verts[f * 9 + 2 * 3 + 2];
    const float xmin = FloatMin3(v0x, v1x, v2x) - sqrt_radius;
    const float xmax = FloatMax3(v0x, v1x, v2x) + sqrt_radius;
    const float ymin = FloatMin3(v0y, v1y, v2y) - sqrt_radius;
    const float ymax = FloatMax3(v0y, v1y, v2y) + sqrt_radius;
    const float zmin = FloatMin3(v0z, v1z, v2z);
    const bool skip = zmin < kEpsilon;
    bboxes[0 * F + f] = xmin;
    bboxes[1 * F + f] = xmax;
    bboxes[2 * F + f] = ymin;
    bboxes[3 * F + f] = ymax;
    skip_face[f] = skip;
  }
}

__global__ void RasterizeCoarseCudaKernel(
    const float* bboxes, // (4, E) (xmin, xmax, ymin, ymax)
    const bool* should_skip, // (E,)
    const int64_t* elem_first_idxs,
    const int64_t* elems_per_batch,
    const int N,
    const int E,
    const int H,
    const int W,
    const int bin_size,
    const int chunk_size,
    const int max_elem_per_bin,
    int* elems_per_bin,
    int* bin_elems) {
  extern __shared__ char sbuf[];
  const int M = max_elem_per_bin;
  // Integer divide round up
  const int num_bins_x = 1 + (W - 1) / bin_size;
  const int num_bins_y = 1 + (H - 1) / bin_size;

  // NDC range depends on the ratio of W/H
  // The shorter side from (H, W) is given an NDC range of 2.0 and
  // the other side is scaled by the ratio of H:W.
  const float NDC_x_half_range = NonSquareNdcRange(W, H) / 2.0f;
  const float NDC_y_half_range = NonSquareNdcRange(H, W) / 2.0f;

  // Size of half a pixel in NDC units is the NDC half range
  // divided by the corresponding image dimension
  const float half_pix_x = NDC_x_half_range / W;
  const float half_pix_y = NDC_y_half_range / H;

  // This is a boolean array of shape (num_bins_y, num_bins_x, chunk_size)
  // stored in shared memory that will track whether each elem in the chunk
  // falls into each bin of the image.
  BitMask binmask((unsigned int*)sbuf, num_bins_y, num_bins_x, chunk_size);

  // Have each block handle a chunk of elements
  const int chunks_per_batch = 1 + (E - 1) / chunk_size;
  const int num_chunks = N * chunks_per_batch;

  for (int chunk = blockIdx.x; chunk < num_chunks; chunk += gridDim.x) {
    const int batch_idx = chunk / chunks_per_batch; // batch index
    const int chunk_idx = chunk % chunks_per_batch;
    const int elem_chunk_start_idx = chunk_idx * chunk_size;

    binmask.block_clear();
    const int64_t elem_start_idx = elem_first_idxs[batch_idx];
    const int64_t elem_stop_idx = elem_start_idx + elems_per_batch[batch_idx];

    // Have each thread handle a different face within the chunk
    for (int e = threadIdx.x; e < chunk_size; e += blockDim.x) {
      const int e_idx = elem_chunk_start_idx + e;

      // Check that we are still within the same element of the batch
      if (e_idx >= elem_stop_idx || e_idx < elem_start_idx) {
        continue;
      }

      if (should_skip[e_idx]) {
        continue;
      }
      const float xmin = bboxes[0 * E + e_idx];
      const float xmax = bboxes[1 * E + e_idx];
      const float ymin = bboxes[2 * E + e_idx];
      const float ymax = bboxes[3 * E + e_idx];

      // Brute-force search over all bins; TODO(T54294966) something smarter.
      for (int by = 0; by < num_bins_y; ++by) {
        // Y coordinate of the top and bottom of the bin.
        // PixToNdc gives the location of the center of each pixel, so we
        // need to add/subtract a half pixel to get the true extent of the bin.
        // Reverse ordering of Y axis so that +Y is upwards in the image.
        const float bin_y_min =
            PixToNonSquareNdc(by * bin_size, H, W) - half_pix_y;
        const float bin_y_max =
            PixToNonSquareNdc((by + 1) * bin_size - 1, H, W) + half_pix_y;
        const bool y_overlap = (ymin <= bin_y_max) && (bin_y_min < ymax);

        for (int bx = 0; bx < num_bins_x; ++bx) {
          // X coordinate of the left and right of the bin.
          // Reverse ordering of x axis so that +X is left.
          const float bin_x_max =
              PixToNonSquareNdc((bx + 1) * bin_size - 1, W, H) + half_pix_x;
          const float bin_x_min =
              PixToNonSquareNdc(bx * bin_size, W, H) - half_pix_x;

          const bool x_overlap = (xmin <= bin_x_max) && (bin_x_min < xmax);
          if (y_overlap && x_overlap) {
            binmask.set(by, bx, e);
          }
        }
      }
    }
    __syncthreads();
    // Now we have processed every elem in the current chunk. We need to
    // count the number of elems in each bin so we can write the indices
    // out to global memory. We have each thread handle a different bin.
    for (int byx = threadIdx.x; byx < num_bins_y * num_bins_x;
         byx += blockDim.x) {
      const int by = byx / num_bins_x;
      const int bx = byx % num_bins_x;
      const int count = binmask.count(by, bx);
      const int elems_per_bin_idx =
          batch_idx * num_bins_y * num_bins_x + by * num_bins_x + bx;

      // This atomically increments the (global) number of elems found
      // in the current bin, and gets the previous value of the counter;
      // this effectively allocates space in the bin_faces array for the
      // elems in the current chunk that fall into this bin.
      const int start = atomicAdd(elems_per_bin + elems_per_bin_idx, count);

      // Now loop over the binmask and write the active bits for this bin
      // out to bin_faces.
      int next_idx = batch_idx * num_bins_y * num_bins_x * M +
          by * num_bins_x * M + bx * M + start;
      for (int e = 0; e < chunk_size; ++e) {
        if (binmask.get(by, bx, e)) {
          // TODO(T54296346) find the correct method for handling errors in
          // CUDA. Throw an error if num_faces_per_bin > max_faces_per_bin.
          // Either decrease bin size or increase max_faces_per_bin
          bin_elems[next_idx] = elem_chunk_start_idx + e;
          next_idx++;
        }
      }
    }
    __syncthreads();
  }
}

at::Tensor RasterizeCoarseCuda(
    const at::Tensor& bboxes,
    const at::Tensor& should_skip,
    const at::Tensor& elem_first_idxs,
    const at::Tensor& elems_per_batch,
    const std::tuple<int, int> image_size,
    const int bin_size,
    const int max_elems_per_bin) {
  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(bboxes.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const int H = std::get<0>(image_size);
  const int W = std::get<1>(image_size);

  const int E = bboxes.size(1);
  const int N = elems_per_batch.size(0);
  const int M = max_elems_per_bin;

  // Integer divide round up
  const int num_bins_y = 1 + (H - 1) / bin_size;
  const int num_bins_x = 1 + (W - 1) / bin_size;

  if (num_bins_y >= kMaxItemsPerBin || num_bins_x >= kMaxItemsPerBin) {
    std::stringstream ss;
    ss << "In RasterizeCoarseCuda got num_bins_y: " << num_bins_y
       << ", num_bins_x: " << num_bins_x << ", "
       << "; that's too many!";
    AT_ERROR(ss.str());
  }
  auto opts = elems_per_batch.options().dtype(at::kInt);
  at::Tensor elems_per_bin = at::zeros({N, num_bins_y, num_bins_x}, opts);
  at::Tensor bin_elems = at::full({N, num_bins_y, num_bins_x, M}, -1, opts);

  if (bin_elems.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return bin_elems;
  }

  const int chunk_size = 512;
  const size_t shared_size = num_bins_y * num_bins_x * chunk_size / 8;
  const size_t blocks = 64;
  const size_t threads = 512;

  RasterizeCoarseCudaKernel<<<blocks, threads, shared_size, stream>>>(
      bboxes.contiguous().data_ptr<float>(),
      should_skip.contiguous().data_ptr<bool>(),
      elem_first_idxs.contiguous().data_ptr<int64_t>(),
      elems_per_batch.contiguous().data_ptr<int64_t>(),
      N,
      E,
      H,
      W,
      bin_size,
      chunk_size,
      M,
      elems_per_bin.data_ptr<int32_t>(),
      bin_elems.data_ptr<int32_t>());

  AT_CUDA_CHECK(hipGetLastError());
  return bin_elems;
}

__global__ void RasterizePointsCoarseCudaKernel(
    const float* points, // (P, 3)
    const int64_t* cloud_to_packed_first_idx, // (N)
    const int64_t* num_points_per_cloud, // (N)
    const float* radius,
    const int N,
    const int P,
    const int H,
    const int W,
    const int bin_size,
    const int chunk_size,
    const int max_points_per_bin,
    int* points_per_bin,
    int* bin_points) {
  extern __shared__ char sbuf[];
  const int M = max_points_per_bin;

  // Integer divide round up
  const int num_bins_x = 1 + (W - 1) / bin_size;
  const int num_bins_y = 1 + (H - 1) / bin_size;

  // NDC range depends on the ratio of W/H
  // The shorter side from (H, W) is given an NDC range of 2.0 and
  // the other side is scaled by the ratio of H:W.
  const float NDC_x_half_range = NonSquareNdcRange(W, H) / 2.0f;
  const float NDC_y_half_range = NonSquareNdcRange(H, W) / 2.0f;

  // Size of half a pixel in NDC units is the NDC half range
  // divided by the corresponding image dimension
  const float half_pix_x = NDC_x_half_range / W;
  const float half_pix_y = NDC_y_half_range / H;

  // This is a boolean array of shape (num_bins_y, num_bins_x, chunk_size)
  // stored in shared memory that will track whether each point in the chunk
  // falls into each bin of the image.
  BitMask binmask((unsigned int*)sbuf, num_bins_y, num_bins_x, chunk_size);

  // Have each block handle a chunk of points and build a 3D bitmask in
  // shared memory to mark which points hit which bins.  In this first phase,
  // each thread processes one point at a time. After processing the chunk,
  // one thread is assigned per bin, and the thread counts and writes the
  // points for the bin out to global memory.
  const int chunks_per_batch = 1 + (P - 1) / chunk_size;
  const int num_chunks = N * chunks_per_batch;
  for (int chunk = blockIdx.x; chunk < num_chunks; chunk += gridDim.x) {
    const int batch_idx = chunk / chunks_per_batch;
    const int chunk_idx = chunk % chunks_per_batch;
    const int point_start_idx = chunk_idx * chunk_size;

    binmask.block_clear();

    // Using the batch index of the thread get the start and stop
    // indices for the points.
    const int64_t cloud_point_start_idx = cloud_to_packed_first_idx[batch_idx];
    const int64_t cloud_point_stop_idx =
        cloud_point_start_idx + num_points_per_cloud[batch_idx];

    // Have each thread handle a different point within the chunk
    for (int p = threadIdx.x; p < chunk_size; p += blockDim.x) {
      const int p_idx = point_start_idx + p;

      // Check if point index corresponds to the cloud in the batch given by
      // batch_idx.
      if (p_idx >= cloud_point_stop_idx || p_idx < cloud_point_start_idx) {
        continue;
      }

      const float px = points[p_idx * 3 + 0];
      const float py = points[p_idx * 3 + 1];
      const float pz = points[p_idx * 3 + 2];
      const float p_radius = radius[p_idx];
      if (pz < 0)
        continue; // Don't render points behind the camera.
      const float px0 = px - p_radius;
      const float px1 = px + p_radius;
      const float py0 = py - p_radius;
      const float py1 = py + p_radius;

      // Brute-force search over all bins; TODO something smarter?
      // For example we could compute the exact bin where the point falls,
      // then check neighboring bins. This way we wouldn't have to check
      // all bins (however then we might have more warp divergence?)
      for (int by = 0; by < num_bins_y; ++by) {
        // Get y extent for the bin. PixToNonSquareNdc gives us the location of
        // the center of each pixel, so we need to add/subtract a half
        // pixel to get the true extent of the bin.
        const float by0 = PixToNonSquareNdc(by * bin_size, H, W) - half_pix_y;
        const float by1 =
            PixToNonSquareNdc((by + 1) * bin_size - 1, H, W) + half_pix_y;
        const bool y_overlap = (py0 <= by1) && (by0 <= py1);

        if (!y_overlap) {
          continue;
        }
        for (int bx = 0; bx < num_bins_x; ++bx) {
          // Get x extent for the bin; again we need to adjust the
          // output of PixToNonSquareNdc by half a pixel.
          const float bx0 = PixToNonSquareNdc(bx * bin_size, W, H) - half_pix_x;
          const float bx1 =
              PixToNonSquareNdc((bx + 1) * bin_size - 1, W, H) + half_pix_x;
          const bool x_overlap = (px0 <= bx1) && (bx0 <= px1);

          if (x_overlap) {
            binmask.set(by, bx, p);
          }
        }
      }
    }
    __syncthreads();
    // Now we have processed every point in the current chunk. We need to
    // count the number of points in each bin so we can write the indices
    // out to global memory. We have each thread handle a different bin.
    for (int byx = threadIdx.x; byx < num_bins_y * num_bins_x;
         byx += blockDim.x) {
      const int by = byx / num_bins_x;
      const int bx = byx % num_bins_x;
      const int count = binmask.count(by, bx);
      const int points_per_bin_idx =
          batch_idx * num_bins_y * num_bins_x + by * num_bins_x + bx;

      // This atomically increments the (global) number of points found
      // in the current bin, and gets the previous value of the counter;
      // this effectively allocates space in the bin_points array for the
      // points in the current chunk that fall into this bin.
      const int start = atomicAdd(points_per_bin + points_per_bin_idx, count);

      // Now loop over the binmask and write the active bits for this bin
      // out to bin_points.
      int next_idx = batch_idx * num_bins_y * num_bins_x * M +
          by * num_bins_x * M + bx * M + start;
      for (int p = 0; p < chunk_size; ++p) {
        if (binmask.get(by, bx, p)) {
          // TODO: Throw an error if next_idx >= M -- this means that
          // we got more than max_points_per_bin in this bin
          // TODO: check if atomicAdd is needed in line 265.
          bin_points[next_idx] = point_start_idx + p;
          next_idx++;
        }
      }
    }
    __syncthreads();
  }
}

at::Tensor RasterizeMeshesCoarseCuda(
    const at::Tensor& face_verts,
    const at::Tensor& mesh_to_face_first_idx,
    const at::Tensor& num_faces_per_mesh,
    const std::tuple<int, int> image_size,
    const float blur_radius,
    const int bin_size,
    const int max_faces_per_bin) {
  TORCH_CHECK(
      face_verts.ndimension() == 3 && face_verts.size(1) == 3 &&
          face_verts.size(2) == 3,
      "face_verts must have dimensions (num_faces, 3, 3)");

  // Check inputs are on the same device
  at::TensorArg face_verts_t{face_verts, "face_verts", 1},
      mesh_to_face_first_idx_t{
          mesh_to_face_first_idx, "mesh_to_face_first_idx", 2},
      num_faces_per_mesh_t{num_faces_per_mesh, "num_faces_per_mesh", 3};
  at::CheckedFrom c = "RasterizeMeshesCoarseCuda";
  at::checkAllSameGPU(
      c, {face_verts_t, mesh_to_face_first_idx_t, num_faces_per_mesh_t});

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(face_verts.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  // Allocate tensors for bboxes and should_skip
  const int F = face_verts.size(0);
  auto float_opts = face_verts.options().dtype(at::kFloat);
  auto bool_opts = face_verts.options().dtype(at::kBool);
  at::Tensor bboxes = at::empty({4, F}, float_opts);
  at::Tensor should_skip = at::empty({F}, bool_opts);

  // Launch kernel to compute triangle bboxes
  const size_t blocks = 128;
  const size_t threads = 256;
  TriangleBoundingBoxKernel<<<blocks, threads, 0, stream>>>(
      face_verts.contiguous().data_ptr<float>(),
      F,
      blur_radius,
      bboxes.contiguous().data_ptr<float>(),
      should_skip.contiguous().data_ptr<bool>());
  AT_CUDA_CHECK(hipGetLastError());

  return RasterizeCoarseCuda(
      bboxes,
      should_skip,
      mesh_to_face_first_idx,
      num_faces_per_mesh,
      image_size,
      bin_size,
      max_faces_per_bin);
}

at::Tensor RasterizePointsCoarseCuda(
    const at::Tensor& points, // (P, 3)
    const at::Tensor& cloud_to_packed_first_idx, // (N)
    const at::Tensor& num_points_per_cloud, // (N)
    const std::tuple<int, int> image_size,
    const at::Tensor& radius,
    const int bin_size,
    const int max_points_per_bin) {
  TORCH_CHECK(
      points.ndimension() == 2 && points.size(1) == 3,
      "points must have dimensions (num_points, 3)");

  // Check inputs are on the same device
  at::TensorArg points_t{points, "points", 1},
      cloud_to_packed_first_idx_t{
          cloud_to_packed_first_idx, "cloud_to_packed_first_idx", 2},
      num_points_per_cloud_t{num_points_per_cloud, "num_points_per_cloud", 3};
  at::CheckedFrom c = "RasterizePointsCoarseCuda";
  at::checkAllSameGPU(
      c, {points_t, cloud_to_packed_first_idx_t, num_points_per_cloud_t});

  // Set the device for the kernel launch based on the device of the input
  at::cuda::CUDAGuard device_guard(points.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  const int H = std::get<0>(image_size);
  const int W = std::get<1>(image_size);

  const int P = points.size(0);
  const int N = num_points_per_cloud.size(0);
  const int M = max_points_per_bin;

  // Integer divide round up.
  const int num_bins_y = 1 + (H - 1) / bin_size;
  const int num_bins_x = 1 + (W - 1) / bin_size;

  if (num_bins_y >= kMaxItemsPerBin || num_bins_x >= kMaxItemsPerBin) {
    // Make sure we do not use too much shared memory.
    std::stringstream ss;
    ss << "In Coarse Rasterizer got num_bins_y: " << num_bins_y
       << ", num_bins_x: " << num_bins_x << ", "
       << "; that's too many!";
    AT_ERROR(ss.str());
  }
  auto opts = num_points_per_cloud.options().dtype(at::kInt);
  at::Tensor points_per_bin = at::zeros({N, num_bins_y, num_bins_x}, opts);
  at::Tensor bin_points = at::full({N, num_bins_y, num_bins_x, M}, -1, opts);

  if (bin_points.numel() == 0) {
    AT_CUDA_CHECK(hipGetLastError());
    return bin_points;
  }

  const int chunk_size = 512;
  const size_t shared_size = num_bins_y * num_bins_x * chunk_size / 8;
  const size_t blocks = 64;
  const size_t threads = 512;

  RasterizePointsCoarseCudaKernel<<<blocks, threads, shared_size, stream>>>(
      points.contiguous().data_ptr<float>(),
      cloud_to_packed_first_idx.contiguous().data_ptr<int64_t>(),
      num_points_per_cloud.contiguous().data_ptr<int64_t>(),
      radius.contiguous().data_ptr<float>(),
      N,
      P,
      H,
      W,
      bin_size,
      chunk_size,
      M,
      points_per_bin.contiguous().data_ptr<int32_t>(),
      bin_points.contiguous().data_ptr<int32_t>());

  AT_CUDA_CHECK(hipGetLastError());
  return bin_points;
}
